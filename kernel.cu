#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <stdio.h>

#define PHASE_VEL   1
#define OUTER_FORSE 10

#define STEP_X 0.5
#define STEP_Y 0.5
#define STEP_T 0.1

static double *hostDataX   = nullptr, *hostDataY   = nullptr, *hostDataZ   = nullptr;
static double *devDataX	   = nullptr, *devDataY    = nullptr, *devDataZ    = nullptr;
static double *devDataBufX = nullptr, *devDataBufY = nullptr, *devDataBufZ = nullptr;

static void _cpuFree() {
	if (::hostDataX)
		std::free((void *)::hostDataX);

	if (::hostDataY)
		std::free((void *)::hostDataY);

	if (::hostDataZ)
		std::free((void *)::hostDataZ);
}

#define cudaCheck
static void _gpuFree() {
	if (::devDataX)
		cudaCheck(hipFree((void *)::devDataX));

	if (::devDataY)
		cudaCheck(hipFree((void *)::devDataY));

	if (::devDataZ)
		cudaCheck(hipFree((void *)::devDataZ));

	if (::devDataBufX)
		cudaCheck(hipFree((void *)::devDataBufX));

	if (::devDataBufY)
		cudaCheck(hipFree((void *)::devDataBufY));

	if (::devDataBufZ)
		cudaCheck(hipFree((void *)::devDataBufZ));
}

/*
* CUDA errors catching block
*/

static void _checkCudaErrorAux(const char *, unsigned, const char *, hipError_t);
#define cudaCheck(value) _checkCudaErrorAux(__FILE__, __LINE__, #value, value)

static void _checkCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;

	std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;

	system("pause");

	_cpuFree();
	_gpuFree();

	exit(1);
}

/*
 * CUDA kernel block
 */

__global__ void kernel(double * __restrict__ z, double * __restrict__ y, double * __restrict__ x,
					   double * __restrict__ yBuf, double * __restrict__ xBuf,
					   const std::size_t size,
					   const double phaseVelocity, const double outerForse,
					   const double stepX, const double stepY,
					   const double stepT, const double maxTime) {
	for (auto t = 0.0; t < maxTime; t += stepT) {
		auto idx = threadIdx.x + blockIdx.x * blockDim.x;

		if (idx < size) {
			xBuf[idx] = (x[idx + 1] + x[idx - 1] + 2.0 * x[idx]) / (stepX * stepX);
			yBuf[idx] = (y[idx + 1] + y[idx - 1] - 2.0 * y[idx]) / (stepY * stepY);
			z[idx]    = (stepT * stepT) * (phaseVelocity * phaseVelocity * (xBuf[idx] + yBuf[idx]) + outerForse);
			printf("%d\n", z[idx]);

			__syncthreads();

			x[idx] = xBuf[idx];
			y[idx] = yBuf[idx];

			__syncthreads();
		}
	}
}

/*
 * Init
 */

int cpuInit(std::size_t size) {
	::hostDataX = (double *)std::calloc(size, sizeof(double));
	if (!::hostDataX)
		return 1;

	::hostDataY = (double *)std::calloc(size, sizeof(double));
	if (!::hostDataY)
		return 1;

	::hostDataZ = (double *)std::calloc(size, sizeof(double));
	if (!::hostDataZ)
		return 1;

	std::memset(::hostDataX, 0, size);
	std::memset(::hostDataY, 0, size);
	std::memset(::hostDataZ, 0, size);

	return 0;
}

void gpuInit(std::size_t size) {
	auto byteSize = size * sizeof(double);

	cudaCheck(hipMalloc((void **)&::devDataX, byteSize));
	cudaCheck(hipMalloc((void **)&::devDataY, byteSize));
	cudaCheck(hipMalloc((void **)&::devDataZ, byteSize));
	cudaCheck(hipMalloc((void **)&::devDataBufX, byteSize));
	cudaCheck(hipMalloc((void **)&::devDataBufY, byteSize));
	cudaCheck(hipMalloc((void **)&::devDataBufZ, byteSize));

	cudaCheck(hipMemset(::devDataX, 0, byteSize));
	cudaCheck(hipMemset(::devDataY, 0, byteSize));
	cudaCheck(hipMemset(::devDataZ, 0, byteSize));
	cudaCheck(hipMemset(::devDataBufX, 0, byteSize));
	cudaCheck(hipMemset(::devDataBufY, 0, byteSize));
	cudaCheck(hipMemset(::devDataBufZ, 0, byteSize));
}

/*
 * Main
 */

int main() {
	const std::size_t size = 100;
	const std::size_t time = 10;

	const auto maxTime = time / STEP_T;

	if (cpuInit(size)) {
		_cpuFree();
		return 1;
	}

	gpuInit(size);

	dim3 nBlocks(1);
	dim3 nThreads(256);

	kernel <<<nBlocks, nThreads>>> (devDataZ, devDataY, devDataX, devDataBufY, devDataBufX, size,
									PHASE_VEL, OUTER_FORSE, STEP_X, STEP_Y, STEP_T, maxTime);

	cudaCheck(hipMemcpy(hostDataZ, devDataZ, size * sizeof(double), hipMemcpyDeviceToHost));

	for (auto i = 0; i < size; i++)
		std::cout << hostDataZ[i] << " ";
	std::cout << std::endl;

	_gpuFree();
	_cpuFree();

	system("pause");

	return 0;
}